#include "hip/hip_runtime.h"
//#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>

#include "compute_kernel.cuh"
#include "compute_alpha.cuh"
#include "compute_obj_robust.cuh"
#include "compute_error_coef.cuh"
#include "utility.h"






int main(int argc, char *argv[])
{
	hipSetDevice(0);
	
	float lambda = static_cast<float>(atof(argv[1]));
	int maxIter = atoi(argv[2]);

	bool verbose = false;
	if (argc == 4)
		verbose = static_cast<bool>(atoi(argv[3]));		

	MATFile *proximal_l0graph_input = matOpen("proximal_l0graph_input.mat","r");
	mxArray *XArray = matGetVariable(proximal_l0graph_input, "X");
	mxArray *l1graph_alphaArray = matGetVariable(proximal_l0graph_input, "alpha");
	mxArray *AArray = matGetVariable(proximal_l0graph_input, "A");
	mxArray *AtAArray = matGetVariable(proximal_l0graph_input, "AtA");
	mxArray *AtXArray = matGetVariable(proximal_l0graph_input, "AtX");
	mxArray *S1Array = matGetVariable(proximal_l0graph_input, "S1");
	mxArray *thrArray = matGetVariable(proximal_l0graph_input, "thr");

	float *h_X = static_cast<float*>(mxGetData(XArray));
	float *h_l1graph_alpha = static_cast<float*>( mxGetData(l1graph_alphaArray));
	float *h_A = static_cast<float*>( mxGetData(AArray));
	float *h_AtA = static_cast<float*>( mxGetData(AtAArray));
	float *h_AtX = static_cast<float*>( mxGetData(AtXArray));
	float S1 = *(static_cast<float*>( mxGetData(S1Array)));
	float thr = *(static_cast<float*>( mxGetData(thrArray)));
	
	

	const mwSize *Xsize = mxGetDimensions(XArray);
	uint d = static_cast<uint>(Xsize[0]);
	uint n = static_cast<uint>(Xsize[1]);
	const mwSize *Asize = mxGetDimensions(AArray);
	uint nA = static_cast<uint>(Asize[1]);	

	float *d_X = NULL, *d_alpha = NULL, *d_A = NULL, *d_AtA = NULL, *d_AtX = NULL;
	float *d_alpha0 = NULL;

	hipMalloc((void**)&d_X,				sizeof(float)*d*n);
	hipMalloc((void**)&d_alpha,			sizeof(float)*nA*n);
	hipMalloc((void**)&d_A,				sizeof(float)*d*nA);
	hipMalloc((void**)&d_AtA,				sizeof(float)*nA*nA);
	hipMalloc((void**)&d_AtX,				sizeof(float)*nA*n);
	hipMalloc((void**)&d_alpha0,			sizeof(float)*nA*n);
	
	
	//debug
	//float *h_alpha = (float*)malloc(sizeof(float)*(nA*n));
	
	
	hipMemcpy(d_X,h_X,sizeof(float)*d*n,hipMemcpyHostToDevice);
	hipMemcpy(d_alpha,h_l1graph_alpha,sizeof(float)*nA*n,hipMemcpyHostToDevice);
	hipMemcpy(d_A,h_A,sizeof(float)*d*nA,hipMemcpyHostToDevice);
	hipMemcpy(d_AtA,h_AtA,sizeof(float)*nA*nA,hipMemcpyHostToDevice);
	hipMemcpy(d_AtX,h_AtX,sizeof(float)*nA*n,hipMemcpyHostToDevice);
	hipMemcpy(d_alpha0,h_l1graph_alpha,sizeof(float)*nA*n,hipMemcpyHostToDevice);

	
	
	matClose(proximal_l0graph_input);


	// create and initialize CUBLAS library object 
	hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
    {
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
            std::cout << "CUBLAS object instantialization error" << std::endl;
        }
        getchar ();
        return 0;
    }

	float elapsedTime = 0;
	
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start, 0);	

	float err = 0.0f;

	for (int iter = 0; iter < maxIter; iter++)
	{
		float c = 2.0f;
		compute_alpha(d_alpha, d_AtA, d_AtX, lambda, c, nA, n, handle);

		//debug
		//hipMemcpy(h_alpha,d_alpha,sizeof(float)*nA*n,hipMemcpyDeviceToHost);
		
		compute_error_coef(&err, d_alpha0, d_alpha, nA, n, handle);

		hipMemcpy(d_alpha0,d_alpha,sizeof(float)*nA*n,hipMemcpyDeviceToHost);


		float obj = 0.0f, l2err = 0.0f, l0_spar_err = 0.0f;
		compute_obj_robust(obj, l2err, l0_spar_err, d_X, d_A, d_alpha, lambda, nA, n, d, handle);

		printf("proximal_manifold: errors = %.5f, iter: %d \n",err,iter);

		if (verbose)
		{
			printf("obj is %.5f, l2err is %.5f, spar_err is %.5f \n", obj,l2err,l0_spar_err);
		}

		/*if (err < thr)
		{
			if (verbose)
			{
				printf("proximal_l0graph converges at iter %d \n", iter);
			}
			break;
		}*/		
		
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
	//hipDeviceSynchronize();
	//finishTime=clock();
	//elapsedTime =(float)(finishTime - startTime);

	// Clean up:
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	printf("time to compute on gpu is %.10f second\n",elapsedTime/(CLOCKS_PER_SEC));




	MATFile *rFile = matOpen("proximal_l0graph_result.mat","w");	
	
	mxArray* alphaoutArray = mxCreateNumericMatrix(nA,n, mxSINGLE_CLASS, mxREAL);
	
	float *h_alphaout   = (float*)mxGetData(alphaoutArray);
	
	//transfer the data from gpu to cpu
	//set Z as the output for its sparsity
	hipMemcpy(h_alphaout,d_alpha,sizeof(float)*nA*n,hipMemcpyDeviceToHost);

	matPutVariable(rFile, "l0graph_alpha", alphaoutArray);
	matClose(rFile);

	mxDestroyArray(alphaoutArray);

	//destroy the input matlab Arrays
	mxDestroyArray(XArray);
	mxDestroyArray(l1graph_alphaArray);
	mxDestroyArray(AArray);
	mxDestroyArray(AtAArray);
	mxDestroyArray(AtXArray);
	mxDestroyArray(S1Array);
	mxDestroyArray(thrArray);


	//deallocation

	//debug
	//free(h_alphai_proximal);

	hipFree(d_X);
	hipFree(d_alpha);
	hipFree(d_A);
	hipFree(d_AtA);
	hipFree(d_AtX);
	hipFree(d_alpha0);
	
	return 0;
}